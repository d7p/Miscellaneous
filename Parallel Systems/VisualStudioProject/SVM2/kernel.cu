#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <types.h>

#define BIAS 0.03662654
#define NUMSUPPORTVECTORS 25

template <unsigned int blockSize>
__global__ void addKernel(double *testVector,double *supportVector1, double *supportVector2,double *alpha, double *result)
{
	//
	__shared__ double temp[NUMSUPPORTVECTORS];
    int Id = (blockIdx.y * blockDim.y + threadIdx.y)+(blockIdx.x * blockDim.x + threadIdx.x);
    temp[Id] = (alpha[Id]*(testVector[1]*supportVector1[Id])+(testVector[2]*supportVector2[Id]))+BIAS;

	//taken from the nivida slides week 3 the 7 version 
	extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;
	sdata[tid] = 0;
	while (i < NUMSUPPORTVECTORS) { sdata[tid] += temp[i] + temp[i+blockSize];  i += gridSize; }
	__syncthreads();
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }
	if (tid < 32) {
		if (blockSize >=  64) sdata[tid] += sdata[tid + 32];
		if (blockSize >=  32) sdata[tid] += sdata[tid + 16];
		if (blockSize >=  16) sdata[tid] += sdata[tid +  8];
		if (blockSize >=   8) sdata[tid] += sdata[tid +  4];
		if (blockSize >=   4) sdata[tid] += sdata[tid +  2];
		if (blockSize >=   2) sdata[tid] += sdata[tid +  1];
	}

	if (tid == 0) result[blockIdx.x] = sdata[0];
}

double SVMCPU(double testVector[2],double supportVector1[NUMSUPPORTVECTORS], double supportVector2[NUMSUPPORTVECTORS],double alpha[NUMSUPPORTVECTORS])
{
	double result=0;
	int i;
	for(i=0;i<NUMSUPPORTVECTORS;i++)
	{
	 result += (alpha[i]*(testVector[1]*supportVector1[i])+(testVector[2]*supportVector2[i]))+BIAS;
	}
	return result;
}

int main()
{
	//host variables
	//I was trying to use a 2d or more array but could not get it to work
	double testVector[2] = {1.552140,1.552510};// if this was to be used test vector should be in input to the main function (supports and alpha could also be inputs)
    double supportVector1[NUMSUPPORTVECTORS] = {1.566690,1.566060,1.566450,1.566800,1.567160,1.566520,1.566630,1.567250,1.566710,1.566750,1.566360,1.566700,1.566640,1.566920,1.566580,1.566650,1.566790,1.566780,1.566760,1.566810,1.566800,1.566820,1.566790,1.566820,1.566980};
    double supportVector2[NUMSUPPORTVECTORS] = {1.566060,1.566450,1.566800,1.567160,1.566520,1.566630,1.567250,1.566710,1.566750,1.566360,1.566700,1.566640,1.566920,1.566580,1.566650,1.566790,1.566780,1.566760,1.566810,1.566800,1.566820,1.566790,1.566820,1.566980,1.566700};
    double alpha[NUMSUPPORTVECTORS] = {-3.128623,3.209386,3.239582,3.229334,-3.123321,3.434454,3.018791,-3.205237,3.490269,-3.324280,3.248263,-3.592508,3.295397,-3.365992,3.466642,3.409259,-3.633737,-3.625539,3.481830,-3.633858,3.506001,-3.617643,3.497924,3.391959,-3.415177};
	double result[NUMSUPPORTVECTORS];
	
	//device variable pointers
	double *testVector_d;
	double *supportVector1_d;
	double *supportVector2_d;
	double *result_d;
	double *alpha_d;
	
	//Timing variables
	float gputime;
	hipEvent_t start;
	hipEvent_t stop;

	//Make space on the device
	hipMalloc((void**)&supportVector1_d, NUMSUPPORTVECTORS*sizeof(double));
	hipMalloc((void**)&supportVector2_d, NUMSUPPORTVECTORS*sizeof(double));
	hipMalloc((void**)&result_d, NUMSUPPORTVECTORS*sizeof(double));
	hipMalloc((void**)&testVector_d, 2*sizeof(double));
	hipMalloc((void**)&alpha_d, NUMSUPPORTVECTORS*sizeof(double));

	//copy host variables
	hipMemcpy(supportVector1_d, supportVector1, NUMSUPPORTVECTORS*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(supportVector2_d, supportVector2, NUMSUPPORTVECTORS*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(result_d, result, NUMSUPPORTVECTORS*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(testVector_d, testVector, 2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(alpha_d, alpha, NUMSUPPORTVECTORS*sizeof(double), hipMemcpyHostToDevice);
	
	//Set the dimentions
	dim3 block(1,1,1);
	dim3 grid(NUMSUPPORTVECTORS,1,1);

	//Start gpu timing
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//Start kernel
	addKernel<32><<<grid,block>>>(testVector_d,supportVector1_d,supportVector2_d,alpha_d,result_d);

	//Copy result back
	hipMemcpy(result, result_d, NUMSUPPORTVECTORS*sizeof(double), hipMemcpyDeviceToHost);

	//Stop gpu timing 
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gputime, start, stop);
	printf("GPU elapesed time:%f\n", gputime);
	
	//free device variables
	hipFree(testVector_d);
	hipFree(supportVector1_d);
	hipFree(supportVector2_d);
	hipFree(result_d);
	hipFree(alpha_d);

	//################CPU########################		
	//Cpu timing start
	struct timeval tp;
	double sec, usec, start, end;

	// Time stamp before the computations
	gettimeofday( &tp, NULL );
	sec = static_cast<double>( tp.tv_sec );
	usec = static_cast<double>( tp.tv_usec )/1E6;
	start = sec + usec;

	// call cpu svm
	double cpuResult = SVMCPU(testVector,supportVector1,supportVector2,alpha);
	
	//Cpu timing stop
	gettimeofday( &tp, NULL );
	sec = static_cast<double>( tp.tv_sec );
	usec = static_cast<double>( tp.tv_usec )/1E6;
	end = sec + usec;

	// Time calculation (in seconds)
	double cputime = end - start;

	//print time
	printf("CPU elapesed time:%d\n",cputime);

	//speed up
	//Scaled size speed up:s+Np
	double ScaledSpeedUp = cputime+(NUMSUPPORTVECTORS*gputime);
	printf("Scaled speed up:%d\n",ScaledSpeedUp);

	//Fixed size speed up: 1/(s+(p/N)
	double FixedSpeedUp = 1/(cputime+(guptime/NUMSUPPORTVECTORS));
	printf("Fixed speed up:%d\n",FixedSpeedUp);

    return 0;

}